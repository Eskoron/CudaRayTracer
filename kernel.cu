
#include "hip/hip_runtime.h"
#include ""

#include "PerspectiveCamera.h"
#include "CudaDefines.h"
#include <stdio.h>
#include <iostream>
#include <string>
#include "Vector3.h"
#include "Rotator.h"
#include "Image.h"
#include "Renderer.h"
#include <GLFW\glfw3.h>


int main()
{


	uint Width = 100;
	uint Height = 100;
	PerspectiveCamera* Cam = new PerspectiveCamera(Vector3(), Rotator(), Width, Height, 45);
	Renderer R = Renderer(Width, Height);





	GLFWwindow* window;

	/* Initialize the library */
	if (!glfwInit())
		return -1;

	/* Create a windowed mode window and its OpenGL context */
	window = glfwCreateWindow(Width, Height, "Hello World", NULL, NULL);
	if (!window)
	{
		glfwTerminate();
		return -1;
	}

	/* Make the window's context current */
	glfwMakeContextCurrent(window);

	glClearColor(1, 0, 0, 1);

	Color* pixels = (Color*)malloc(Width*Height*sizeof(Color));

	glEnable(GL_TEXTURE_2D);
	GLuint texID;
	glGenTextures(1, &texID);
	glBindTexture(GL_TEXTURE_2D, texID);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_REPEAT);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_REPEAT);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
	glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA8, Width, Height, 0, GL_RGBA, GL_UNSIGNED_BYTE, 0);

	/* Loop until the user closes the window */

	int windowWidth;
	int windowHeight;
	glfwGetWindowSize(window, &windowWidth, &windowHeight);

	int Amount_Spheres = 3;

	Sphere* spheres;
	spheres = (Sphere*)malloc(Amount_Spheres * sizeof(Sphere));

	spheres[0].Position = Vector3(10, 0, 0);
	spheres[0].Radius = 5;

	spheres[1].Position = Vector3(0, 10, 0);
	spheres[1].Radius = 5;

	spheres[2].Position = Vector3(0, 0, 10);
	spheres[2].Radius = 5;

	while (!glfwWindowShouldClose(window))
	{
		float r = (float)((double)rand() / (RAND_MAX)) + 1;
		for (uint i = 0; i < Width*Height; i++){

			pixels[i].R = (uchar)255;
			pixels[i].G = (uchar)255;
			pixels[i].B = (uchar)255;
			pixels[i].A = (uchar)255;
		}



		R.Render(Cam, spheres, Amount_Spheres, pixels);

		hipDeviceSynchronize();


		glClearColor(0, 0, 0, 1);
		glClear(GL_COLOR_BUFFER_BIT);

		/* Render here */
		glfwGetWindowSize(window, &windowWidth, &windowHeight);

		glViewport(0, 0, windowWidth, windowHeight);
		glMatrixMode(GL_PROJECTION);
		glLoadIdentity();
		glOrtho(0, 1, 0, 1, 0, 1);
		glMatrixMode(GL_MODELVIEW);





		glBindTexture(GL_TEXTURE_2D, texID);
		glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA8, Width, Height, 0, GL_RGBA, GL_UNSIGNED_BYTE, pixels);
		//glColor3f(0, 1, 0);
		glBegin(GL_QUADS);
		glTexCoord2f(0.0f, 0.0f);
		glVertex2f(-1, -1);
		glTexCoord2f(1.0f, 0.0f);
		glVertex2f(1, -1);
		glTexCoord2f(1.0f, 1.0f);
		glVertex2f(1, 1);
		glTexCoord2f(0.0f, 1.0f);
		glVertex2f(-1, 1);
		glEnd();

		//glDrawPixels(Width,Height,GL_RGBA8,GL_UNSIGNED_BYTE,pixels);

		/* Swap front and back buffers */
		glfwSwapBuffers(window);

		/* Poll for and process events */
		glfwPollEvents();
	}

	free(pixels);
	delete Cam;

	glfwTerminate();
	return 0;

}
