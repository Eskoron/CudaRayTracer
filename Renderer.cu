#include "hip/hip_runtime.h"
#include "Renderer.h"



kernel void GetSphereIntersectionKernel(const Ray* D_Rays, const Sphere* D_Spheres, const uint spheres, const uint Width, const uint Height, Color* Device_Color){

	const int2 thread_2D_pos = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
		blockIdx.y * blockDim.y + threadIdx.y);

	if (thread_2D_pos.x >= Width || thread_2D_pos.y >= Height){
		return;
	}

	//1D-Position In Image
	const int thread_1D_pos = thread_2D_pos.y * Width + thread_2D_pos.x;
	//float Dist = FLT_MAX;
	//for (uint i = 0; i < spheres; i++){
		//D_Spheres[i].intersectRay(D_Rays[thread_1D_pos], Dist);
	//}

	Device_Color[thread_1D_pos].A = 0xFF;
	Device_Color[thread_1D_pos].B = 0x00;
	Device_Color[thread_1D_pos].G = 0xFF;
	Device_Color[thread_1D_pos].R = 0x00;
	//if (Dist < FLT_MAX){
		//Device_Color[thread_1D_pos].G = 0x00;
	//}


}