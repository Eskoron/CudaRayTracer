#include "hip/hip_runtime.h"
#include "Renderer.h"
kernel void GetSphereIntersectionKernel(const Ray* D_Rays, const Sphere* D_Spheres, const uint spheres, const float Width, const float Height, Color* C){

	const int2 thread_2D_pos = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
		blockIdx.y * blockDim.y + threadIdx.y);

	if (thread_2D_pos.x >= Width || thread_2D_pos.y >= Height){
		return;
	}

	//1D-Position In Image
	const int thread_1D_pos = thread_2D_pos.y * Width + thread_2D_pos.x;
	float Dist = FLT_MAX;
	for (uint i = 0; i < spheres; i++){
		D_Spheres[i].intersectRay(D_Rays[thread_1D_pos], Dist);
	}

	C[thread_1D_pos].A = 255.0f;
	C[thread_1D_pos].B = 255.0f;
	C[thread_1D_pos].G = 255.0f;

	if (Dist < FLT_MAX){
		C[thread_1D_pos].R = 255.0f;
	}


}